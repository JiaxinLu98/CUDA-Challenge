// Tiled Duration: 0.1780 ms


#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>

#define FILTER_RADIUS 3
#define IN_TILE_DIM 32
#define OUT_TILE_DIM (IN_TILE_DIM - 2 * FILTER_RADIUS)

__constant__ int mask[2 * FILTER_RADIUS + 1][2 * FILTER_RADIUS + 1];

__global__ void convolution_2d(int *matrix, int *result, int width, int height) {
    // Global thread ID calculation
    int row = blockIdx.y * OUT_TILE_DIM + threadIdx.y - FILTER_RADIUS;
    int col = blockIdx.x * OUT_TILE_DIM + threadIdx.x - FILTER_RADIUS;

    // Load input tile (static)
    __shared__ int s_matrix[IN_TILE_DIM][IN_TILE_DIM];

    if(row >= 0 && row < height && col >= 0 && col < width) {
        s_matrix[threadIdx.y][threadIdx.x] = matrix[row * width + col];
    }
    else {
        s_matrix[threadIdx.y][threadIdx.x] = 0;
    }
    __syncthreads();

    // Calculate output elements
    int tileRow = threadIdx.y - FILTER_RADIUS;
    int tileCol = threadIdx.x - FILTER_RADIUS;

    // Turn off the threads at the edges of the block
    if(row >= 0 && row < height && col >= 0 && col < width) {
        if(tileRow >= 0 && tileRow < OUT_TILE_DIM && tileCol >= 0 && tileCol < OUT_TILE_DIM) {
            int temp = 0;

            // Go over elements in mask
            for(int i = 0; i < 2 * FILTER_RADIUS + 1; i++) {
                for(int j = 0; j < 2 * FILTER_RADIUS + 1; j++) {
                    temp += s_matrix[tileRow + i][tileCol + j] * mask[i][j];
                }
            }

            result[row * width + col] = temp;
        }
    }
}

// Warmup Kernel
__global__
void warmupKernel(){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx == 0){
        printf("CUDA Warmup Done!\n");
    }
}

void verify_result(int *matrix, int *mask, int *result, int N, int mask_dim) {
    for (int row = 0; row < N; row++) {
        for (int col = 0; col < N; col++) {
            int temp = 0;

            for (int i = 0; i < mask_dim; i++) {
                for (int j = 0; j < mask_dim; j++) {
                    int inRow = row - FILTER_RADIUS + i;
                    int inCol = col - FILTER_RADIUS + j;

                    if (inRow >= 0 && inRow < N && inCol >= 0 && inCol < N) {
                        temp += mask[i * mask_dim + j] * matrix[inRow * N + inCol];
                    }
                }
            }

            // Use assert to validate each result
            assert(result[row * N + col] == temp);
        }
    }
}

int main() {
    // Dimensions of the matrix (2 ^ 10 x 2 ^ 10)
    int N = 1 << 10;

    // Size of the matrix (in bytes)
    size_t bytes_n = N * N * sizeof(int);

    // Size of the mask in bytes
    int mask_dim = 2 * FILTER_RADIUS + 1;
    size_t bytes_m = mask_dim * mask_dim * sizeof(int);

    // Allocate the matrix, result matrix, and mask ...
    int *h_matrix = new int[N * N];
    int *h_result = new int[N * N];
    int *h_mask = new int[mask_dim * mask_dim];

    // ... initialize the matrix and the mask
    for(int i = 0; i < N; i++) {
        for(int j = 0; j < N; j++) {
            h_matrix[i * N + j] = rand() % 100;
        }
    }

    for(int i = 0; i < mask_dim; i++) {
        for(int j = 0; j < mask_dim; j++) {
            h_mask[i * mask_dim + j] = rand() % 10;
        }
    }

    // Allocate device memory
    int *d_matrix, *d_result;
    hipMalloc(&d_matrix, bytes_n);
    hipMalloc(&d_result, bytes_n);

    // Copy data to the device
    hipMemcpy(d_matrix, h_matrix, bytes_n, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(mask), h_mask, bytes_m);

    // Define time measure
    double iStart, iElaps;
    hipDeviceSynchronize();
    
    // **Run warmupKernel once to remove first-run overhead**
    iStart = clock();
    warmupKernel<<<1, 1>>>();
    hipDeviceSynchronize();
    iElaps = (clock() - iStart) / CLOCKS_PER_SEC;
    printf("warmup elapsed %.6f sec \n", iElaps);

    // Calculate grid dimensions
    int THREADS = IN_TILE_DIM;
    int BLOCKS = (N + OUT_TILE_DIM - 1) / OUT_TILE_DIM;

    // Dimension launch arguments
    dim3 block_dim(THREADS, THREADS);
    dim3 grid_dim(BLOCKS, BLOCKS);

    // Start to measure runtime
    iStart = clock();

    // Perform 2D Convolution
    convolution_2d<<<grid_dim, block_dim>>>(d_matrix, d_result, N, N);

    hipDeviceSynchronize();
    iElaps = ((clock() - iStart) / CLOCKS_PER_SEC) * 1000;
    printf("Tiled Duration: %.4f ms \n", iElaps);

    // Copy the result back to the CPU
    hipMemcpy(h_result, d_result, bytes_n, hipMemcpyDeviceToHost);

    // Functional test
    verify_result(h_matrix, h_mask, h_result, N, mask_dim);

    std::cout << "COMPLETED SUCCESSFULLY!" << std::endl;

    // Free the memory we allocated
    delete[] h_matrix;
    delete[] h_result;
    delete[] h_mask;

    hipFree(d_matrix);
    hipFree(d_result);

    return 0;
}