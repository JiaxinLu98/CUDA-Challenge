
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include <ctime>

__global__ void convolution_1d(int *array, int *mask, int *result, int n, int m) {
    // Global thread ID calculation
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate radius of the mask
    int r = m / 2;

    // Calculate the starting point for the element
    int start = tid - r;

    // Temp value for calculation
    int temp = 0;

    // Go over each element of the mask
    for(int i = 0; i < m; i++) {
        // Ignore elements that hang off (0s don't contribute)
        if(((start + i) >= 0) && ((start + i) < n)) {
            temp += array[start + i] * mask[i];
        }
    }

    // Write-back the results
    result[tid] = temp;
}



// Verify the result on the CPU
void verify_result(int *array, int *mask, int *result, int n, int m) {
    int radius = m / 2;
    int temp;
    int start;
    for (int i = 0; i < n; i++) {
      start = i - radius;
      temp = 0;
      for (int j = 0; j < m; j++) {
        if ((start + j >= 0) && (start + j < n)) {
          temp += array[start + j] * mask[j];
        }
      }
      assert(temp == result[i]);
    }
}

int main() {
    // Number of elements in result array
    int n = 1 << 20;

    // Size of the array in bytes
    int bytes_n = n * sizeof(int);

    // Number of elements in the convolution mask
    int m = 7;

    // Size of mask in bytes
    int bytes_m = m * sizeof(int);

    // Allocate the array (include edge elements)...
    int *h_array = new int[n];

    // ... and initialize it
    for(int i = 0; i < n; i++) {
        h_array[i] = rand() % 100;
    }

    // Allocate the mask and initialize it
    int *h_mask = new int[m];
    for(int i = 0; i < m; i++) {
        h_mask[i] = rand() % 10;
    }

    // Allocate the result array
    int *h_result = new int[n];

    // Allocate space on the device
    int *d_array, *d_mask, *d_result;
    hipMalloc(&d_array, bytes_n);
    hipMalloc(&d_mask, bytes_m);
    hipMalloc(&d_result, bytes_n);

    // Copy the data to the device
    hipMemcpy(d_array, h_array, bytes_n, hipMemcpyHostToDevice);
    hipMemcpy(d_mask, h_mask, bytes_m, hipMemcpyHostToDevice);

    // Threads per block
    int THREADS = 256;

    // Number of blocks
    int GRID = (n + THREADS - 1) / THREADS;

    // Call the kernel
    convolution_1d<<<GRID, THREADS>>>(d_array, d_mask, d_result, n, m);

    // Copy back the result
    hipMemcpy(h_result, d_result, bytes_n, hipMemcpyDeviceToHost);

    // Verify the result
    verify_result(h_array, h_mask, h_result, n, m);

    std::cout << "COMPLETED SUCCESSFULLY" << std::endl;

    return 0;
}
